
#include <hip/hip_runtime.h>
__global__ void bedSlopeSourceSolver(float *BedSlopeSource, float *U, float *BottomIntPts, int m, int n, float dx, float dy)
{
	// Calculate the row and column of the thread within the thread block
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	// First check if the thread is operating on a cell inside of the block's one cell deep ghost cells
	if (col > 0 && row > 0 && col < n-1 && row < m-1)
	{
		// Calculate index of this cell in the BedSlopeSource matrix
		int slopeIndex = row*n*2 + col*2;
		
		// Calculate index of this cell in the U matrix
		int uIndex = row*n*3 + col*3;
		
		// Calculate index of this cell in the BottomIntPts matrix
		int bottomIndex = row*(n+1)*2 + col*2;
		
		// Calculate the water depth at the center of the cell
		// Note: h = w - B;
		//	 w is the water column height at the cell center measured from z = 0;
		//	 B is the terrain elevation measured from z = 0, and is calculated as the average
		//	   elevation of the center of two opposing edges (in this case, left and right)
		float hCenter =  U[uIndex] - (BottomIntPts[bottomIndex+1] + BottomIntPts[bottomIndex + 1*2 + 1])/2.0f;
		
		// Calculate the slope of the terrain in both x- and y-directions
		float slopeX = (BottomIntPts[bottomIndex + 3] - BottomIntPts[bottomIndex + 1]) / dx;
		float slopeY = (BottomIntPts[bottomIndex + (n+1)*2] - BottomIntPts[bottomIndex]) / dy;
		
		// Calculate the bed slope source terms
		BedSlopeSource[slopeIndex] = -9.81f * slopeX * hCenter;
		BedSlopeSource[slopeIndex+1] = -9.81f * slopeY * hCenter;		 
	}
}


__global__ void bedShearSourceSolver(float *BedShearSource, float *U, float *BottomIntPts, int m, int n, float dx, float dy)
{
	// Define constants used in calculating shear friction and velocity
	float manningsN = 0.03f;
	float sqrt2 = sqrtf(2.0f);
	float Kappa = 0.01f * fmaxf(1.0f, fminf(dx, dy));
	
	// Calculate the row and column of the thread within the thread block
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	// First check if the thread is operating on a cell inside of the block's one cell deep ghost cells
	if (col > 0 && row > 0 && col < n-1 && row < m-1)
	{
		// Calculate index of cell in U
		int uIndex = row*n*3 + col*3;
		
		// Calculate indices of right and left interface elevations
		int leftBottomIndex = row*(n+1)*2 + col*2 + 1;
		int rightBottomIndex = leftBottomIndex + 2;
		
		// Water column height is w-B
		float h = U[uIndex] - (BottomIntPts[leftBottomIndex] + BottomIntPts[rightBottomIndex]) / 2.0f;
		if (h > 0.0f)
		{
			// Calculate Chezy Friction Coefficient from Manning's N
			float Cz = powf(h, 1.0f/6.0f)/manningsN;
			
			// Calculatate u and v using the damping method for shoal zones (see Brodtkorb eq. 5)
			float denom = sqrtf(powf(h, 4.0f) + fmaxf(powf(h, 4.0f), Kappa));
			float u = (sqrt2 * h * U[uIndex+1]) / denom;
			float v = (sqrt2 * h * U[uIndex+2]) / denom;
			
			// Calculate bed shear
			BedShearSource[row*n + col] = (-9.81f * sqrtf(powf(u, 2.0f) + powf(v, 2.0f))) / (h * powf(Cz, 2.0f));
		} else {
			// There is no water, so there is no bed shear
			BedShearSource[row*n + col] = 0.0f;
		}
	}
}
